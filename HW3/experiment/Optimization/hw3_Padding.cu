#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#define US_PER_SEC 1000000
#define BS 32
#define LBS 5 // 1<<LBS = BS

const int INF = 1073741823;
void input(char *inFileName);
void output(char *outFileName);
void block_FW(int B);
__global__ void phase1(int B, int r, int *Dist_GPU, int n);
__global__ void phase2(int B, int r, int *Dist_GPU, int n);
__global__ void phase3(int B, int r, int *Dist_GPU, int n);

int n, m, n_origin;
int *Dist, *Dist_GPU;

int main(int argc, char* argv[]){
    struct timeval start, end;
    double time;
    gettimeofday(&start, NULL);

    input(argv[1]);
    int B = BS;
    block_FW(B);
    output(argv[2]);

    gettimeofday(&end, NULL);
    time = (double)(US_PER_SEC*(end.tv_sec-start.tv_sec)+(end.tv_usec-start.tv_usec))/US_PER_SEC;
    printf("Time: %.2lf\n", time);
    return 0;
}

void input(char *inFileName){
    FILE *file = fopen(inFileName, "rb");
    fread(&n, sizeof(int), 1, file);
    fread(&m, sizeof(int), 1, file);

    n_origin = n;
    n += BS-((n%BS+BS-1)%BS+1);
    hipHostMalloc(&Dist, n*n*sizeof(int), hipHostMallocDefault);
    for(int i = 0; i < n; i++){
        for(int j = 0; j < n; j++){
            Dist[i*n+j] = (i==j&&i<n_origin)?0:INF;
        }
    }

    int pair[3];
    for(int i = 0; i < m; i++){
        fread(pair, sizeof(int), 3, file);
        Dist[pair[0]*n+pair[1]] = pair[2];
    }
    fclose(file);
}

void output(char *outFileName){
    FILE *file = fopen(outFileName, "w");
    for(int i = 0; i < n_origin; i++){
        fwrite(&Dist[i*n], sizeof(int), n_origin, file);
    }
    fclose(file);
    hipHostFree(Dist);
}

void block_FW(int B){
    hipMalloc(&Dist_GPU, n*n*sizeof(int));
    hipMemcpy(Dist_GPU, Dist, n*n*sizeof(int), hipMemcpyHostToDevice);
    int round = n/BS;
    for(int r = 0; r < round; r++){
        phase1 <<<1, dim3(BS, BS)>>> (BS, r, Dist_GPU, n);
        phase2 <<<dim3(2, round-1), dim3(BS, BS)>>> (BS, r, Dist_GPU, n);
        phase3 <<<dim3(round-1, round-1), dim3(BS, BS)>>> (BS, r, Dist_GPU, n);
    }
    hipMemcpy(Dist, Dist_GPU, n*n*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(Dist_GPU);
}

__global__ void phase1(int B, int r, int *Dist_GPU, int n){
    int b_i = r<<LBS, b_j = r<<LBS, b_k = r<<LBS;
    int i = threadIdx.x, j = threadIdx.y;

    for(int k = 0; k < BS; k++){
        __syncthreads();
        Dist_GPU[(b_i+i)*n+(b_j+j)] = min(Dist_GPU[(b_i+i)*n+(b_j+j)], Dist_GPU[(b_i+i)*n+(b_k+k)]+Dist_GPU[(b_k+k)*n+(b_j+j)]);
    }
}

__global__ void phase2(int B, int r, int *Dist_GPU, int n){
    // ROW: (blockIdx.x = 1), COL: (blockIdx.y = 0)
    int b_i = (blockIdx.x*r+(!blockIdx.x)*(blockIdx.y+(blockIdx.y>=r)))<<LBS;
    int b_j = (blockIdx.x*(blockIdx.y+(blockIdx.y>=r))+(!blockIdx.x)*r)<<LBS;
    int b_k = r<<LBS;
    int i = threadIdx.x, j = threadIdx.y;

    __syncthreads();
    for(int k = 0; k < BS; k++){
        Dist_GPU[(b_i+i)*n+(b_j+j)] = min(Dist_GPU[(b_i+i)*n+(b_j+j)], Dist_GPU[(b_i+i)*n+(b_k+k)]+Dist_GPU[(b_k+k)*n+(b_j+j)]);
    }
}

__global__ void phase3(int B, int r, int *Dist_GPU, int n){
    int b_i = (blockIdx.x+(blockIdx.x>=r))<<LBS, b_j = (blockIdx.y+(blockIdx.y>=r))<<LBS, b_k = r<<LBS;
    int i = threadIdx.x, j = threadIdx.y;

    __syncthreads();
    for(int k = 0; k < BS; k++){
        Dist_GPU[(b_i+i)*n+(b_j+j)] = min(Dist_GPU[(b_i+i)*n+(b_j+j)], Dist_GPU[(b_i+i)*n+(b_k+k)]+Dist_GPU[(b_k+k)*n+(b_j+j)]);
    }
}